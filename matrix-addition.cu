#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>


void allocateCPUMemory(float **h_A, float** h_B, float** h_C, int size){

  printf("CPU memory allocating ...\n");
  *h_A =(float*) malloc(sizeof(float)*size*size);
  *h_B =(float*) malloc(sizeof(float)*size*size);
  *h_C =(float*) malloc(sizeof(float)*size*size);

  printf("CPU memory allocated...\n");

}

void allocateGPUMemory(float **d_A, float** d_B, float** d_C, int size){

   hipMalloc(((void**)d_A),  sizeof(float)*size*size);
   hipMalloc(((void**)d_B),  sizeof(float)*size*size);
   hipMalloc(((void**)d_C),  sizeof(float)*size*size);

}

void initCPUData(float* h_A, float* h_B, int size){

  for(int i =0; i < size; i++){
    for(int j=0; j < size; j++){
      int offset = i*size+j;
      // printf("offset: %d\n", offset);
      *(h_A+offset) = offset;
      *(h_B+offset) = 2*offset;
    }
  }
}

void printdata(float* x, int size){

  for(int i =0; i < size; i++){
    for(int j=0; j< size; j++){
      int offset = i*size+j;
      printf("%f ", x[offset]);
    }
    printf("\n");
  }

}


__global__ void AddKernel(float* d_A, float* d_B, float* d_C, int size){

  // printf("GPU ...........");
  int row = threadIdx.y + blockIdx.y*blockDim.y ;
  int col = threadIdx.x + blockIdx.x*blockDim.x ;
  // printf("blockDim.x: %d, blockIdx.x: %d , threadIdx.x:%d \n", blockDim.x, blockIdx.x, threadIdx.x);
  // printf("blockDim.y: %d, blockIdx.y: %d , threadIdx.y:%d \n", blockDim.y, blockIdx.y, threadIdx.y);
  int offset = row*size + col;
  
  if(row < size && col < size){
    printf("threadIdx.x:%d threadIdx.y:%d ---> row: %d col:%d\n", threadIdx.x, threadIdx.y, row, col);
    // printf("Kernel played for : row: %d col: %d \n", row, col);
    // printf("row: %d , col %d, idx: %d\n", row, col, offset);
    d_C[offset]= d_A[offset] + d_B[offset];
  }
}

__global__ void AddPerRowKernel(float* d_A, float* d_B, float* d_C, int size){

  int row = threadIdx.y + blockIdx.y*blockDim.y ;
  // int col = threadIdx.x + blockIdx.x*blockDim.x ;

  if(row < size){
    printf("Kernel played for : row: %d\n", row);
    for(int col =0; col < size; col++){
      int offset = row*size + col;
      d_C[offset] = d_A[offset] + d_B[offset];
    }
  }
}

__global__  void printData(float* d_A){
  int size = 4;
  int row = threadIdx.y + blockIdx.y*blockDim.y;
  int col = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = row*size + col;
  if(row < size && col < size){
    printf("offset: %d , value: %f", offset, d_A[offset]);

  } 
} 

double cpuSecond() {
 struct timeval tp;
 gettimeofday(&tp,NULL);
 return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


int main(){

  float* h_A, *d_A;
  float* h_B, *d_B;
  float* h_C, *d_C;
  int size = 4; 

  int nElements = size*size;

  allocateCPUMemory(&h_A, &h_B, &h_C, size);
  allocateGPUMemory(&d_A, &d_B, &d_C, size);

  // cudaMalloc(((void**)&d_A),  sizeof(float)*size*size);
  //  cudaMalloc(((void**)&d_B),  sizeof(float)*size*size);
  //  cudaMalloc(((void**)&d_C),  sizeof(float)*size*size);

  printf("allocation done\n");
  initCPUData(h_A, h_B, size);

  // printf(" ------------------ Data A --------------------------\n");
  // printdata(h_A, size);
  // printf(" ------------------ Data B --------------------------\n");
  // printdata(h_B, size);

  hipMemcpy(d_A, h_A, nElements*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, nElements*sizeof(float), hipMemcpyHostToDevice);

  // // kernel call
  dim3 block(32,32,1);
  dim3 grid(size+block.x-1/block.x,size+block.y-1/block.y,1);
   
  AddKernel<<<grid,block>>>(d_A, d_B, d_C, size);
  // AddPerRowKernel<<<grid,block>>>(d_A, d_B, d_C, size);
  // printData<<<1,block>>>(d_A);
  hipDeviceSynchronize();

  // // copying result back;
  hipMemcpy(h_C, d_C, nElements*sizeof(float), hipMemcpyDeviceToHost);

  // printf(" ------------------ Data C --------------------------\n");
  // printdata(h_C, size);

  //verify data
  bool veri_pass = true;
  for(int i =0; i < size ; i++){
    for(int j =0; j < size; j++){
      int idx = i*size + j;
      if(h_C[idx] != (h_A[idx] + h_B[idx])){
        veri_pass = false;
        break;
      }
    }
    if(!veri_pass) break;
  }

  if(veri_pass){
    printf("Verification pass .......");
  }
  else {
    printf("verification failed .......");
  }

  hipFree(d_A);
  hipFree(d_B);

  free(h_A);
  free(h_B);
  free(h_C);


  hipDeviceReset();

  return 0;
}